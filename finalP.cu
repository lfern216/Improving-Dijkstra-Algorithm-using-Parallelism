#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <cstdlib>
#include <stdio.h>
#include <limits.h>
#include <sys/time.h>

using namespace std;

const int NUMTHREADS = 128;
 
// Number of vertices in the graph
#define V 1024

// Find the vertex with minimum distance value, from
// the set of vertices NOT yet included in updater[]
//the result would override updater[]  

int minDistance(int shortPath[], bool updater[]) {
   // Initialize min value
   int min = INT_MAX, min_index;

   for (int x = 0; x < V; x++)
     if (updater[x] == false && shortPath[x] <= min){
         min = shortPath[x], min_index = x;
}
	//printf("%d Min: ",min_index);
   return min_index;
}//end of method


void printSolution(int shortPath[], int n) {
   printf("Vertex Distance from Source\n");
   for (int i = 0; i < V; i++)
      printf("%d \t\t %d\n", i, shortPath[i]);
} //end of method

// Funtion that implements Dijkstra's single source shortest path algorithm
// for a graph represented using adjacency matrix representation

__global__ void updateDistance(bool* updater, long* matrix, int* shortPath, int u){

int i = blockIdx.x * blockDim.x + threadIdx.x;

         // checking for the shortest distance and updating shortPath
         // Update shortPath[i] only when updater[i]!=true, there is an edge from  
         // the program will enter the if statement  as long as there are smaller nodes than the current one

         if(i<V){

    if ((!updater[i] && matrix[u*V+i] && shortPath[u] != INT_MAX && shortPath[u]+matrix[u*V+i] < shortPath[i]))
            shortPath[i] = shortPath[u] + matrix[u*V+i];

      }//end of if(big)
 
}//end of method

long* generateAdjMatrix(int count){

long* randoNumber = (long*)malloc(count*count*sizeof(long));

srand(time(NULL));

for(int i=0;i<count;i++){
	for(int j=0;j<count;j++){
		if(i !=j){
			long randomResult = rand()%2;			
			randoNumber[(i*count)+j] = randomResult;
			randoNumber[(j*count)+i]= randomResult;
		}
	}
}//end of for
	return randoNumber;
}//end of method

void printMatrix(int count, long* matrix){
//count is the size of the matrix

	for(int i=0;i<count;i++){

	         for(int j=0;j<count;j++){
	         printf("\t%3ld", matrix[(i*count)+j]);
		}
	printf("\n");
	}
}//end of method

int main() {

     //GPU
     int *d_shortPath;
     bool *d_updater;
     long* d_matrix;

     //allocate CPU variables in memory
     bool* updater = (bool *)malloc(V*sizeof(bool));
     int* shortPath = (int *)malloc(V*sizeof(int));
     long* matrix = (long *)malloc(V*V*sizeof(long));

     //generate adj matrix
     matrix = generateAdjMatrix(V);

     //print Matrix
     //printMatrix(V,matrix);

     // Initialize all distances as INFINITE(Maximum integer value) and stpSet[] as false
     for (int i = 0; i < V; i++){
        shortPath[i] = INT_MAX, updater[i] = false;
      }

     // Distance of source vertex from itself is always 0
     shortPath[0] = 0;

     //allocate GPU variables in memory
     hipMalloc((void**) &d_updater,(V*sizeof(bool)));
     hipMalloc((void**) &d_shortPath,(V*sizeof(int)));
     hipMalloc((void**) &d_matrix,(V*V*sizeof(long)));

     //copying the matrix values to the GPU
     hipMemcpy(d_matrix,matrix,V*V*sizeof(long),hipMemcpyHostToDevice);

     //declaring the clock variables
     struct timeval start, end;

     // Find shortest path for all vertices
     for (int count = 0; count < V-1; count++) {
     // For every iteration, a minimum distance vertex is chosen from the set of vertices not yet processed
     //u is always equal to 0 in first iteration.
     int u = minDistance(shortPath, updater);	

     //start the clock
     gettimeofday(&start,NULL);
	
     // Mark the picked vertex as processed
     updater[u] = true;

     //copy the updates to GPU so we can call updateDistance 	
     hipMemcpy(d_updater,updater,V*sizeof(bool),hipMemcpyHostToDevice);
     hipMemcpy(d_shortPath,shortPath,V*sizeof(int),hipMemcpyHostToDevice);

     //call updatedistance, 1st parameter=# of blocks we have,2nd= #of threads in that block
     updateDistance<<<V,NUMTHREADS>>>(d_updater,d_matrix,d_shortPath,u);
	
     //copy the updated values to the CPU
     hipMemcpy(shortPath,d_shortPath,V*sizeof(int),hipMemcpyDeviceToHost);
     hipMemcpy(updater,d_updater,V*sizeof(bool),hipMemcpyDeviceToHost);
      
     }//end of for

     printf("\n");

     //stop clock
     gettimeofday(&end,0); //stop timing
      
     // print the constructed distance array
     printSolution(shortPath, V);

     printf("+++++++++");

     //calculate total time
     double elapsed = (end.tv_usec - start.tv_usec); // * 1000 + (end.tv_sec - start.tv_sec) * 1000;
     printf("Time elapsed: %f ms\n", elapsed);

     //free memory space
     free(updater);
     free(shortPath);
     free(matrix);

     //free GPU variables
     hipFree(d_shortPath);
     hipFree(d_matrix);
     hipFree(d_updater);    
    
     return 0;
}//end of main
